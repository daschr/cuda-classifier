#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <string.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#include "ls_cl.h"
#include "parser.h"
}

#define RINGBUF_SIZE 32
#define RINGBUF_MASK 31

static inline void check_error(hipError_t e, const char *file, int line) {
    if(e != hipSuccess) {
        fprintf(stderr, "[ERROR] %s in %s (line %d)\n", hipGetErrorString(e), file, line);
        exit(EXIT_FAILURE);
    }
}
#define CHECK(X) (check_error(X, __FILE__, __LINE__))

static inline void cpy_rules(const ruleset_t *rules, uint32_t *buffer, uint8_t upper) {
    size_t bp;
    for(size_t i=0; i<rules->num_rules; ++i) {
        bp=i<<2;
        buffer[bp++]=rules->rules[i].c1[upper];
        buffer[bp++]=rules->rules[i].c2[upper];
        buffer[bp++]=(uint32_t) (rules->rules[i].c3[upper]<<16) | (uint32_t) rules->rules[i].c4[upper];
        buffer[bp]=rules->rules[i].c5[upper];
    }
}

void *get_results(void *p) {
    ls_cl_t *lscl=(ls_cl_t *) p;
    bool stream_running;
    int32_t pos;
    for(size_t i=0;; i=(i+1)&RINGBUF_MASK) {
        do {
            pthread_mutex_lock(&lscl->running_mtxs[i]);
            stream_running=lscl->streams_running[i];
            pthread_mutex_unlock(&lscl->running_mtxs[i]);
            if(!lscl->running&&!stream_running) goto end;
        } while(!stream_running);

        hipStreamSynchronize(lscl->streams[i]);
        pos=*lscl->pos_ring_h[i];

        fprintf(lscl->outfile, "%02X\n", pos==UINT_MAX?0xff:lscl->ruleset->rules[pos].val);
        *lscl->pos_ring_h[i]=UINT_MAX;

        pthread_mutex_lock(&lscl->running_mtxs[i]);
        lscl->streams_running[i]=0;
        pthread_mutex_unlock(&lscl->running_mtxs[i]);
    }
end:
    return NULL;
}

__global__ void ls(	const __restrict__ uint *lower, const __restrict__ uint *upper, const ulong rules_size,
                    const __restrict__ uint *header, uint *pos) {

    ulong start=blockDim.x*blockIdx.x+threadIdx.x, step=(gridDim.x*blockDim.x)<<2;
    __shared__ uint8_t found;
    __shared__ uint h[4];
    ulong i;
    uint8_t r;

    if(!threadIdx.x) {
        found=0;
#pragma unroll
        for(int i=0; i<4; ++i)
            h[i]=header[i];
        __threadfence_block();
    }

    __syncthreads();
    i=start<<2;
    while(!found) {
        r=i<rules_size?lower[i]<=h[0] & h[0]<=upper[i]
          & lower[i+1]<=h[1] & h[1]<=upper[i+1]
          & (__vcmpleu2(lower[i+2], h[2]) & __vcmpgeu2(upper[i+2], h[2]))==0xffffffff
          & lower[i+3]<=h[3] & h[3]<=upper[i+3]:0;

        if(r) {
            atomicMin((uint *) pos, i>>2);
            found=1;
            __threadfence_system();
        }

        if((!start) & (i>rules_size))
            found=1;

        i+=step;
        __syncthreads();
    }
}

bool ls_cl_new(ls_cl_t *lscl, const ruleset_t *rules, FILE *outfile) {
    lscl->ruleset=rules;
    lscl->streams_running=(uint8_t *) malloc(sizeof(uint8_t)*RINGBUF_SIZE);
    memset(lscl->streams_running, 0, sizeof(uint8_t)*RINGBUF_SIZE);
    lscl->running=1;
    lscl->outfile=outfile;

    lscl->running_mtxs=(pthread_mutex_t *) malloc(sizeof(pthread_mutex_t)*RINGBUF_SIZE);
    for(size_t i=0; i<RINGBUF_SIZE; ++i)
        lscl->running_mtxs[i]=PTHREAD_MUTEX_INITIALIZER;

    // lower upper buffer

    size_t bufsize=(sizeof(uint32_t)<<2)*rules->num_rules;
    uint32_t *buffer=(uint32_t *) malloc(bufsize);
    memset(buffer, 0, bufsize);

    CHECK(hipMalloc((void **) &lscl->lower, bufsize));
    CHECK(hipMalloc((void **) &lscl->upper, bufsize));

    cpy_rules(rules, buffer, 0);
    CHECK(hipMemcpy(lscl->lower, buffer, bufsize, hipMemcpyHostToDevice));

    cpy_rules(rules, buffer, 1);
    CHECK(hipMemcpy(lscl->upper, buffer, bufsize, hipMemcpyHostToDevice));

    // head pos ring buffer
    lscl->pos_ring_h=(uint32_t **) malloc(sizeof(uint32_t *)*RINGBUF_SIZE);
    lscl->pos_ring=(uint32_t **) malloc(sizeof(uint32_t *)*RINGBUF_SIZE);
    lscl->header_ring_h=(uint32_t **) malloc(sizeof(uint32_t *)*RINGBUF_SIZE);
    lscl->header_ring=(uint32_t **) malloc(sizeof(uint32_t *)*RINGBUF_SIZE);


    for(size_t i=0; i<RINGBUF_SIZE; ++i) {
        CHECK(hipHostAlloc((void **) &(lscl->header_ring_h[i]), (sizeof(uint32_t)<<2), hipHostMallocMapped));
        CHECK(hipHostGetDevicePointer((void **) &(lscl->header_ring[i]), lscl->header_ring_h[i], 0));
        CHECK(hipHostAlloc((void **) &(lscl->pos_ring_h[i]), sizeof(uint32_t), hipHostMallocMapped));
        CHECK(hipHostGetDevicePointer((void **) &(lscl->pos_ring[i]), lscl->pos_ring_h[i], 0));
    }

    lscl->streams=(hipStream_t *) malloc(sizeof(hipStream_t)*RINGBUF_SIZE);
    for(size_t i=0; i<RINGBUF_SIZE; ++i)
        CHECK(hipStreamCreateWithFlags(lscl->streams+i, 0));

    CHECK(hipDeviceGetAttribute(&lscl->mp_count, hipDeviceAttributeMultiprocessorCount, 0));

    pthread_create(&lscl->getrest, NULL, get_results, (void *) lscl);

    free(buffer);

    return true;
}

void ls_cl_get(ls_cl_t *lscl, const header_t *header) {
    static uint32_t i=0;

    lscl->header_ring_h[i][0]=header->h1;
    lscl->header_ring_h[i][1]=header->h2;
    lscl->header_ring_h[i][2]=((uint32_t) header->h3<<16)|(uint32_t) header->h4;
    lscl->header_ring_h[i][3]=header->h5;

    ls<<<1,128,0,lscl->streams[i]>>>(lscl->lower, lscl->upper, (uint64_t) lscl->ruleset->num_rules<<2,
                                     lscl->header_ring[i], lscl->pos_ring[i]);

    uint8_t stream_running;
    do {
        pthread_mutex_lock(&lscl->running_mtxs[i]);
        stream_running=lscl->streams_running[i];
        pthread_mutex_unlock(&lscl->running_mtxs[i]);
    } while(stream_running);


    pthread_mutex_lock(&lscl->running_mtxs[i]);
    lscl->streams_running[i]=1;
    pthread_mutex_unlock(&lscl->running_mtxs[i]);

    i=(i+1)&RINGBUF_MASK;
}

void ls_cl_free(ls_cl_t *lscl) {
    //lscl->running=0;
    //pthread_join(lscl->getrest, NULL);
    hipFree(lscl->lower);
    hipFree(lscl->upper);
    for(size_t i=0; i<RINGBUF_SIZE; ++i) {
        hipHostFree(lscl->pos_ring_h[i]);
        hipHostFree(lscl->header_ring_h[i]);
    }
    free(lscl->pos_ring);
    free(lscl->header_ring);
    free(lscl->pos_ring_h);
    free(lscl->header_ring_h);
    free(lscl->streams_running);
    free(lscl->streams);
    free(lscl->running_mtxs);
}

int main(int ac, char *as[]) {
    if(ac<3) {
        fprintf(stderr, "Usage: %s [ruleset] [headers] [?result file]\n", as[0]);
        return EXIT_FAILURE;
    }

    FILE *res_file=stdout;
    if(ac>3) {
        if((res_file=fopen(as[3], "w"))==NULL) {
            fprintf(stderr, "could not open \"%s\" for writing!\n", as[3]);
            return EXIT_FAILURE;
        }
    }

    ruleset_t rules= {.num_rules=0, .rules_size=0, .rules=NULL};
    headers_t headers= {.num_headers=0, .headers_size=0, .headers=NULL};
    if(!parse_ruleset(&rules, as[1]) || !parse_headers(&headers, as[2]))
        goto fail;

    struct timeval tv1, tv2;
    ls_cl_t lscl;

    gettimeofday(&tv1, NULL);
    if(!ls_cl_new(&lscl, &rules, res_file)) {
        fputs("could not initiate ls_cl!\n", stderr);
        goto fail;
    }
    gettimeofday(&tv2, NULL);
    printf("PREPROCESSING  took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    gettimeofday(&tv1, NULL);
    for(size_t i=0; i<headers.num_headers; ++i)
        ls_cl_get(&lscl, headers.headers+i);
    lscl.running=0;
    pthread_join(lscl.getrest, NULL);
    gettimeofday(&tv2, NULL);
    printf("CLASSIFICATION took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    ls_cl_free(&lscl);

    return EXIT_SUCCESS;
fail:
    free(rules.rules);
    free(headers.headers);

    return EXIT_FAILURE;
}
