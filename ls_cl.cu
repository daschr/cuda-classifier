#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <string.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#include "ls_cl.h"
#include "parser.h"
}

static inline void check_error(hipError_t e, const char *file, int line) {
    if(e != hipSuccess) {
        fprintf(stderr, "[ERROR] %s in %s (line %d)\n", hipGetErrorString(e), file, line);
        exit(EXIT_FAILURE);
    }
}
#define CHECK(X) (check_error(X, __FILE__, __LINE__))

static inline void cpy_rules(const ruleset_t *rules, uint32_t *buffer, uint8_t upper) {
    for(size_t i=0; i<rules->num_rules; ++i) {
        buffer[i<<3]=rules->rules[i].c1[upper];
        buffer[(i<<3)+1]=rules->rules[i].c2[upper];
        buffer[(i<<3)+2]=rules->rules[i].c3[upper];
        buffer[(i<<3)+3]=rules->rules[i].c4[upper];
        buffer[(i<<3)+4]=rules->rules[i].c5[upper];
    }
}

__global__ void ls(	uint *lower, uint *upper, ulong num_rules, volatile uint *header, uint *pos,
                    volatile unsigned char *new_pkt, volatile unsigned char *done_pkt, volatile unsigned char *running) {
    uint start=(uint) blockDim.x*blockIdx.x+threadIdx.x, step=(uint) gridDim.x*blockDim.x;
    
	ulong bp;
    unsigned char r;
    while(*running) {
        if(start==0) {
			while(*new_pkt==0);
            *new_pkt=0;
        }

		__threadfence();

        for(uint i=start; i<num_rules; i+=step) {
            bp=i<<3;
            r= lower[bp]<=header[0] & header[0]<=upper[bp];
            ++bp;
            r&=lower[bp]<=header[1] & header[1]<=upper[bp];
            ++bp;
            r&=lower[bp]<=header[2] & header[2]<=upper[bp];
            ++bp;
            r&=lower[bp]<=header[3] & header[3]<=upper[bp];
            ++bp;
            r&=lower[bp]<=header[4] & header[4]<=upper[bp];
            if(r) {
                atomicMin(pos, i);
                break;
            }
        }

		if(start==0){
			*done_pkt=1;
		}
    }
}

bool ls_cl_new(ls_cl_t *lscl, const ruleset_t *rules) {
    size_t bufsize=(sizeof(uint32_t)<<3)*rules->num_rules;
    uint32_t *buffer=(uint32_t *) malloc(bufsize);
    memset(buffer, 0, bufsize);
    CHECK(hipMalloc((void **) &lscl->lower, bufsize));
    CHECK(hipMalloc((void **) &lscl->upper, bufsize));

    CHECK(hipHostAlloc((void **) &lscl->header_h, sizeof(uint32_t)<<3, hipHostMallocMapped));
    CHECK(hipHostAlloc((void **) &lscl->pos_h, sizeof(uint32_t), hipHostMallocMapped));
    CHECK(hipHostAlloc((void **) &lscl->new_pkt_h, sizeof(unsigned char), hipHostMallocMapped));
    CHECK(hipHostAlloc((void **) &lscl->done_pkt_h, sizeof(unsigned char), hipHostMallocMapped));
    CHECK(hipHostAlloc((void **) &lscl->running_h, sizeof(unsigned char), hipHostMallocMapped));

    CHECK(hipHostGetDevicePointer((void **) &lscl->header, lscl->header_h, 0));
    CHECK(hipHostGetDevicePointer((void **) &lscl->pos, lscl->pos_h, 0));
    CHECK(hipHostGetDevicePointer((void **) &lscl->new_pkt, lscl->new_pkt_h, 0));
    CHECK(hipHostGetDevicePointer((void **) &lscl->done_pkt, lscl->done_pkt_h, 0));
    CHECK(hipHostGetDevicePointer((void **) &lscl->running, lscl->running_h, 0));

    cpy_rules(rules, buffer, 0);
    CHECK(hipMemcpy(lscl->lower, buffer, bufsize, hipMemcpyHostToDevice));

    cpy_rules(rules, buffer, 1);
    CHECK(hipMemcpy(lscl->upper, buffer, bufsize, hipMemcpyHostToDevice));

	hipStream_t stream;
	CHECK(hipStreamCreateWithFlags(&stream, 0));
	
	*lscl->running=1;
	ls<<<1,1024,0,stream>>>(lscl->lower, lscl->upper, (uint64_t) rules->num_rules, lscl->header, lscl->pos, lscl->new_pkt, lscl->done_pkt, lscl->running);

    free(buffer);

    return true;
}

uint8_t ls_cl_get(ls_cl_t *lscl, const ruleset_t *rules, const header_t *header) {
	static const struct timespec ts={.tv_sec=0, .tv_nsec=10};
#define H(X) lscl->header_h[X-1]=header->h ## X
    H(1);
    H(2);
    H(3);
    H(4);
    H(5);
#undef H
    *lscl->pos_h=UINT_MAX;
	*lscl->new_pkt_h=1;
	*lscl->done_pkt_h=0;

	while(!(*lscl->done_pkt_h)) nanosleep(&ts, NULL);

	return *lscl->pos_h==UINT_MAX?0xff:rules->rules[*lscl->pos_h].val;
}

void ls_cl_free(ls_cl_t *lscl) {
	printf("stopping...");
	*lscl->running_h=0;
	printf("stopping...");
	hipFree(lscl->lower);
    hipFree(lscl->upper);
    hipHostFree(lscl->pos_h);
    hipHostFree(lscl->header_h);
}

int main(int ac, char *as[]) {
    if(ac<3) {
        fprintf(stderr, "Usage: %s [ruleset] [headers] [?result file]\n", as[0]);
        return EXIT_FAILURE;
    }

    FILE *res_file=stdout;
    if(ac>3) {
        if((res_file=fopen(as[3], "w"))==NULL) {
            fprintf(stderr, "could not open \"%s\" for writing!\n", as[3]);
            return EXIT_FAILURE;
        }
    }

    ruleset_t rules= {.num_rules=0, .rules_size=0, .rules=NULL};
    headers_t headers= {.num_headers=0, .headers_size=0, .headers=NULL};
    if(!parse_ruleset(&rules, as[1]) || !parse_headers(&headers, as[2]))
        goto fail;

    struct timeval tv1, tv2;
    ls_cl_t lscl;

    gettimeofday(&tv1, NULL);
    if(!ls_cl_new(&lscl, &rules)) {
        fputs("could not initiate ls_cl!\n", stderr);
        goto fail;
    }
    gettimeofday(&tv2, NULL);
    printf("PREPROCESSING  took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    gettimeofday(&tv1, NULL);
    for(size_t i=0; i<headers.num_headers; ++i)
        fprintf(res_file, "%02X\n", ls_cl_get(&lscl, &rules, headers.headers+i));
    gettimeofday(&tv2, NULL);
    printf("CLASSIFICATION took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    ls_cl_free(&lscl);

    return EXIT_SUCCESS;
fail:
    free(rules.rules);
    free(headers.headers);

    return EXIT_FAILURE;
}
