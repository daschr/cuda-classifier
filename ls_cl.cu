#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <string.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#include "ls_cl.h"
#include "parser.h"
}

static inline void check_error(hipError_t e, const char *file, int line) {
    if(e != hipSuccess) {
        fprintf(stderr, "[ERROR] %s in %s (line %d)\n", hipGetErrorString(e), file, line);
        exit(EXIT_FAILURE);
    }
}
#define CHECK(X) (check_error(X, __FILE__, __LINE__))

static inline void cpy_rules(const ruleset_t *rules, uint32_t *buffer, uint8_t upper) {
    for(size_t i=0; i<rules->num_rules; ++i) {
        buffer[i<<3]=rules->rules[i].c1[upper];
        buffer[(i<<3)+1]=rules->rules[i].c2[upper];
        buffer[(i<<3)+2]=rules->rules[i].c3[upper];
        buffer[(i<<3)+3]=rules->rules[i].c4[upper];
        buffer[(i<<3)+4]=rules->rules[i].c5[upper];
    }
}

__global__ void ls(uint *lower, uint *upper, ulong num_rules, uint *header, uint *pos) {
    uint start=(uint) blockDim.x*blockIdx.x+threadIdx.x, step=(uint) gridDim.x*blockDim.x;
	ulong bp;
    unsigned char r;
	for(uint i=start; i<num_rules; i+=step) {
        bp=i<<3;
		r= lower[bp]<=header[0] & header[0]<=upper[bp]; ++bp;
        r&=lower[bp]<=header[1] & header[1]<=upper[bp]; ++bp;
        r&=lower[bp]<=header[2] & header[2]<=upper[bp]; ++bp;
        r&=lower[bp]<=header[3] & header[3]<=upper[bp]; ++bp;
        r&=lower[bp]<=header[4] & header[4]<=upper[bp]; ++bp;
        if(r) {
            atomicMin(pos, i);
            break;
        }
    }
}

bool ls_cl_new(ls_cl_t *lscl, const ruleset_t *rules) {
    size_t bufsize=(sizeof(uint32_t)<<3)*rules->num_rules;
    uint32_t *buffer=(uint32_t *) malloc(bufsize);
    memset(buffer, 0, bufsize);
    CHECK(hipMalloc((void **) &lscl->lower, bufsize));
    CHECK(hipMalloc((void **) &lscl->upper, bufsize));
	
	CHECK(hipHostAlloc((void **) &lscl->header_h, sizeof(uint32_t)<<3, hipHostMallocMapped));
    CHECK(hipHostAlloc((void **) &lscl->pos_h, sizeof(uint64_t), hipHostMallocMapped));

	CHECK(hipHostGetDevicePointer((void **) &lscl->header, lscl->header_h, 0));
	CHECK(hipHostGetDevicePointer((void **) &lscl->pos, lscl->pos_h, 0));

	cpy_rules(rules, buffer, 0);
    CHECK(hipMemcpy(lscl->lower, buffer, bufsize, hipMemcpyHostToDevice));

    cpy_rules(rules, buffer, 1);
    CHECK(hipMemcpy(lscl->upper, buffer, bufsize, hipMemcpyHostToDevice));

    free(buffer);

    return true;
}

uint8_t ls_cl_get(ls_cl_t *lscl, const ruleset_t *rules, const header_t *header) {
#define H(X) lscl->header_h[X-1]=header->h ## X
	H(1); H(2); H(3); H(4); H(5);
#undef H
	lscl->pos_h[0]=UINT_MAX;

    ls<<<512,512>>>(lscl->lower, lscl->upper, (uint64_t) rules->num_rules, lscl->header, lscl->pos);
	hipDeviceSynchronize();

    return lscl->pos_h[0]==UINT_MAX?0xff:rules->rules[lscl->pos[0]].val;
}

void ls_cl_free(ls_cl_t *lscl) {
    hipFree(lscl->lower);
    hipFree(lscl->upper);
    hipHostFree(lscl->pos_h);
    hipHostFree(lscl->header_h);
}

int main(int ac, char *as[]) {
    if(ac<3) {
        fprintf(stderr, "Usage: %s [ruleset] [headers] [?result file]\n", as[0]);
        return EXIT_FAILURE;
    }

    FILE *res_file=stdout;
    if(ac>3) {
        if((res_file=fopen(as[3], "w"))==NULL) {
            fprintf(stderr, "could not open \"%s\" for writing!\n", as[3]);
            return EXIT_FAILURE;
        }
    }

    ruleset_t rules= {.num_rules=0, .rules_size=0, .rules=NULL};
    headers_t headers= {.num_headers=0, .headers_size=0, .headers=NULL};
    if(!parse_ruleset(&rules, as[1]) || !parse_headers(&headers, as[2]))
        goto fail;

    struct timeval tv1, tv2;
    ls_cl_t lscl;

    gettimeofday(&tv1, NULL);
    if(!ls_cl_new(&lscl, &rules)) {
        fputs("could not initiate ls_cl!\n", stderr);
        goto fail;
    }
    gettimeofday(&tv2, NULL);
    printf("PREPROCESSING  took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    gettimeofday(&tv1, NULL);
    for(size_t i=0; i<headers.num_headers; ++i)
        fprintf(res_file, "%02X\n", ls_cl_get(&lscl, &rules, headers.headers+i));
    gettimeofday(&tv2, NULL);
    printf("CLASSIFICATION took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    ls_cl_free(&lscl);

    return EXIT_SUCCESS;
fail:
    free(rules.rules);
    free(headers.headers);

    return EXIT_FAILURE;
}
