#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <string.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#include "ls_cl.h"
#include "parser.h"
}

static inline void check_error(hipError_t e, const char *file, int line) {
    if(e != hipSuccess) {
        fprintf(stderr, "[ERROR] %s in %s (line %d)\n", hipGetErrorString(e), file, line);
        exit(EXIT_FAILURE);
    }
}
#define CHECK(X) (check_error(X, __FILE__, __LINE__))

static inline void cpy_rules(const ruleset_t *rules, uint32_t *buffer, uint8_t upper) {
    for(size_t i=0; i<rules->num_rules; ++i) {
        buffer[i<<3]=rules->rules[i].c1[upper];
        buffer[(i<<3)+1]=rules->rules[i].c2[upper];
        buffer[(i<<3)+2]=rules->rules[i].c3[upper];
        buffer[(i<<3)+3]=rules->rules[i].c4[upper];
        buffer[(i<<3)+4]=rules->rules[i].c5[upper];
    }
}

__global__ void ls(	uint *lower, uint *upper, ulong num_rules, volatile uint *header, volatile uint *pos,
                    volatile uint *new_pkt, volatile uint *done_pkt, volatile uint32_t *running) {

    uint start=(uint) blockDim.x*blockIdx.x+threadIdx.x, step=(uint) gridDim.x*blockDim.x;
    ulong bp;
    unsigned char r;
    for(; *running;) {
        while(*new_pkt==0);

        for(uint i=start; i<num_rules; i+=step) {
            bp=i<<3;
            r= lower[bp]<=header[0] & header[0]<=upper[bp];
            ++bp;
            r&=lower[bp]<=header[1] & header[1]<=upper[bp];
            ++bp;
            r&=lower[bp]<=header[2] & header[2]<=upper[bp];
            ++bp;
            r&=lower[bp]<=header[3] & header[3]<=upper[bp];
            ++bp;
            r&=lower[bp]<=header[4] & header[4]<=upper[bp];
            if(r) {
                atomicMin((uint *)pos, i);
                break;
            }
        }
        __syncthreads();
        __threadfence();
        if(start==0) {
            *new_pkt=0;
            *done_pkt=1;
            __threadfence_system();
        }
    }
}
bool ls_cl_new(ls_cl_t *lscl, const ruleset_t *rules) {
    // lower upper buffer

    size_t bufsize=(sizeof(uint32_t)<<3)*rules->num_rules;
    uint32_t *buffer=(uint32_t *) malloc(bufsize);
    memset(buffer, 0, bufsize);

    CHECK(hipMalloc((void **) &lscl->lower, bufsize));
    CHECK(hipMalloc((void **) &lscl->upper, bufsize));

    cpy_rules(rules, buffer, 0);
    CHECK(hipMemcpy(lscl->lower, buffer, bufsize, hipMemcpyHostToDevice));

    cpy_rules(rules, buffer, 1);
    CHECK(hipMemcpy(lscl->upper, buffer, bufsize, hipMemcpyHostToDevice));

    free(buffer);

    CHECK(hipHostAlloc((void **) &lscl->header_h, (sizeof(uint32_t)<<3), hipHostMallocMapped));
    CHECK(hipHostGetDevicePointer((void **) &lscl->header, lscl->header_h, 0));
    CHECK(hipHostAlloc((void **) &lscl->pos_h, sizeof(uint32_t), hipHostMallocMapped));
    CHECK(hipHostGetDevicePointer((void **) &lscl->pos, lscl->pos_h, 0));

    CHECK(hipHostAlloc((void **) &lscl->new_pkt_h, sizeof(uint32_t), hipHostMallocMapped));
    CHECK(hipHostGetDevicePointer((void **) &lscl->new_pkt, (uint32_t *) lscl->new_pkt_h, 0));
    CHECK(hipHostAlloc((void **) &lscl->done_pkt_h, sizeof(uint32_t), hipHostMallocMapped));
    CHECK(hipHostGetDevicePointer((void **) &lscl->done_pkt, (uint32_t *) lscl->done_pkt_h, 0));
    CHECK(hipHostAlloc((void **) &lscl->running_h, sizeof(uint32_t), hipHostMallocMapped));
    CHECK(hipHostGetDevicePointer((void **) &lscl->running, (uint32_t *) lscl->running_h, 0));

    *lscl->new_pkt_h=0;
    *lscl->done_pkt_h=0;
    *lscl->running_h=1;

    CHECK(hipStreamCreateWithFlags(&lscl->kernel_stream, 0));

    int mp_count;
    CHECK(hipDeviceGetAttribute(&mp_count, hipDeviceAttributeMultiprocessorCount, 0));
    ls<<<mp_count, 256, 0,lscl->kernel_stream>>>(lscl->lower, lscl->upper, (uint64_t) rules->num_rules,
            lscl->header, lscl->pos, lscl->new_pkt, lscl->done_pkt, lscl->running);

    return true;
}

uint8_t ls_cl_get(ls_cl_t *lscl, const header_t *header, const ruleset_t *rules) {
#define H(X) lscl->header_h[X-1]=header->h ## X
    H(1);
    H(2);
    H(3);
    H(4);
    H(5);
#undef H

    *lscl->pos_h=UINT_MAX;
    *lscl->new_pkt_h=1;
    while(!(*lscl->done_pkt_h));
    *lscl->done_pkt_h=0;

    return *lscl->pos_h==UINT_MAX?0xff:rules->rules[*lscl->pos_h].val;
}

void ls_cl_free(ls_cl_t *lscl) {
    *lscl->running_h=0;
    *lscl->new_pkt_h=1;
    hipStreamSynchronize(lscl->kernel_stream);
    CHECK(hipFree(lscl->lower));
    CHECK(hipFree(lscl->upper));
    CHECK(hipHostFree(lscl->header_h));
    CHECK(hipHostFree(lscl->pos_h));
    CHECK(hipHostFree((void *) lscl->done_pkt_h));
    CHECK(hipHostFree((void *) lscl->new_pkt_h));
    CHECK(hipHostFree((void *) lscl->running_h));
}

int main(int ac, char *as[]) {
    if(ac<3) {
        fprintf(stderr, "Usage: %s [ruleset] [headers] [?result file]\n", as[0]);
        return EXIT_FAILURE;
    }

    FILE *res_file=stdout;
    if(ac>3) {
        if((res_file=fopen(as[3], "w"))==NULL) {
            fprintf(stderr, "could not open \"%s\" for writing!\n", as[3]);
            return EXIT_FAILURE;
        }
    }

    ruleset_t rules= {.num_rules=0, .rules_size=0, .rules=NULL};
    headers_t headers= {.num_headers=0, .headers_size=0, .headers=NULL};
    if(!parse_ruleset(&rules, as[1]) || !parse_headers(&headers, as[2]))
        goto fail;

    struct timeval tv1, tv2;
    ls_cl_t lscl;

    gettimeofday(&tv1, NULL);
    if(!ls_cl_new(&lscl, &rules)) {
        fputs("could not initiate ls_cl!\n", stderr);
        goto fail;
    }
    gettimeofday(&tv2, NULL);
    printf("PREPROCESSING  took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    gettimeofday(&tv1, NULL);
    for(size_t i=0; i<headers.num_headers; ++i)
        fprintf(res_file, "%02X\n", ls_cl_get(&lscl, headers.headers+i, &rules));
    gettimeofday(&tv2, NULL);
    printf("CLASSIFICATION took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    ls_cl_free(&lscl);

    return EXIT_SUCCESS;
fail:
    free(rules.rules);
    free(headers.headers);

    return EXIT_FAILURE;
}
