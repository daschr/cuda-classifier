#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <string.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#include "ls_cl.h"
#include "parser.h"
}

#define RINGBUF_SIZE 32
#define RINGBUF_MASK 31

static inline void check_error(hipError_t e, const char *file, int line) {
    if(e != hipSuccess) {
        fprintf(stderr, "[ERROR] %s in %s (line %d)\n", hipGetErrorString(e), file, line);
        exit(EXIT_FAILURE);
    }
}
#define CHECK(X) (check_error(X, __FILE__, __LINE__))

static inline void cpy_rules(const ruleset_t *rules, uint32_t *buffer, uint8_t upper) {
    for(size_t i=0; i<rules->num_rules; ++i) {
        buffer[i<<3]=rules->rules[i].c1[upper];
        buffer[(i<<3)+1]=rules->rules[i].c2[upper];
        buffer[(i<<3)+2]=rules->rules[i].c3[upper];
        buffer[(i<<3)+3]=rules->rules[i].c4[upper];
        buffer[(i<<3)+4]=rules->rules[i].c5[upper];
    }
}

void *get_results(void *p) {
    ls_cl_t *lscl=(ls_cl_t *) p;
    bool stream_running;
    int32_t pos;
    for(size_t i=0;; i=(i+1)&RINGBUF_MASK) {
        do {
            pthread_mutex_lock(&lscl->running_mtxs[i]);
            stream_running=lscl->streams_running[i];
            pthread_mutex_unlock(&lscl->running_mtxs[i]);
            if(!lscl->running&&!stream_running) goto end;
        } while(!stream_running);

        hipStreamSynchronize(lscl->streams[i]);
        pos=*lscl->pos_ring_h[i];

        fprintf(lscl->outfile, "%02X\n", pos==UINT_MAX?0xff:lscl->ruleset->rules[pos].val);
        *lscl->pos_ring_h[i]=UINT_MAX;

        pthread_mutex_lock(&lscl->running_mtxs[i]);
        lscl->streams_running[i]=0;
        pthread_mutex_unlock(&lscl->running_mtxs[i]);
    }
end:
    return NULL;
}

__global__ void ls(	const __restrict__ uint *lower, const __restrict__ uint *upper, const ulong num_rules, const __restrict__ uint *header, uint *pos) {

    uint start=(uint) blockDim.x*blockIdx.x+threadIdx.x, step=(uint) gridDim.x*blockDim.x;
    ulong bp;
    __shared__ uint8_t found;

    if(!threadIdx.x){
        found=0;
	}
    __syncthreads();

    for(uint i=start; i<num_rules; i+=step) {
        bp=i<<3;
        if(lower[bp]<=header[0] & header[0]<=upper[bp]
                & lower[bp+1]<=header[1] & header[1]<=upper[bp+1]
                & lower[bp+2]<=header[2] & header[2]<=upper[bp+2]
                & lower[bp+3]<=header[3] & header[3]<=upper[bp+3]
                & lower[bp+4]<=header[4] & header[4]<=upper[bp+4]) {
            atomicMin(pos, i);
            found=1;
            __threadfence_system();
        }
        if(found)
            break;
    }
}

bool ls_cl_new(ls_cl_t *lscl, const ruleset_t *rules, FILE *outfile) {
    lscl->ruleset=rules;
    lscl->streams_running=(uint8_t *) malloc(sizeof(uint8_t)*RINGBUF_SIZE);
    memset(lscl->streams_running, 0, sizeof(uint8_t)*RINGBUF_SIZE);
    lscl->running=1;
    lscl->outfile=outfile;

    lscl->running_mtxs=(pthread_mutex_t *) malloc(sizeof(pthread_mutex_t)*RINGBUF_SIZE);
    for(size_t i=0; i<RINGBUF_SIZE; ++i)
        lscl->running_mtxs[i]=PTHREAD_MUTEX_INITIALIZER;

    // lower upper buffer

    size_t bufsize=(sizeof(uint32_t)<<3)*rules->num_rules;
    uint32_t *buffer=(uint32_t *) malloc(bufsize);
    memset(buffer, 0, bufsize);

    CHECK(hipMalloc((void **) &lscl->lower, bufsize));
    CHECK(hipMalloc((void **) &lscl->upper, bufsize));

    cpy_rules(rules, buffer, 0);
    CHECK(hipMemcpy(lscl->lower, buffer, bufsize, hipMemcpyHostToDevice));

    cpy_rules(rules, buffer, 1);
    CHECK(hipMemcpy(lscl->upper, buffer, bufsize, hipMemcpyHostToDevice));

    // head pos ring buffer
    lscl->pos_ring_h=(uint32_t **) malloc(sizeof(uint32_t *)*RINGBUF_SIZE);
    lscl->pos_ring=(uint32_t **) malloc(sizeof(uint32_t *)*RINGBUF_SIZE);
    lscl->header_ring_h=(uint32_t **) malloc(sizeof(uint32_t *)*RINGBUF_SIZE);
    lscl->header_ring=(uint32_t **) malloc(sizeof(uint32_t *)*RINGBUF_SIZE);


    for(size_t i=0; i<RINGBUF_SIZE; ++i) {
        CHECK(hipHostAlloc((void **) &(lscl->header_ring_h[i]), (sizeof(uint32_t)<<3), hipHostMallocMapped));
        CHECK(hipHostGetDevicePointer((void **) &(lscl->header_ring[i]), lscl->header_ring_h[i], 0));
        CHECK(hipHostAlloc((void **) &(lscl->pos_ring_h[i]), sizeof(uint32_t), hipHostMallocMapped));
        CHECK(hipHostGetDevicePointer((void **) &(lscl->pos_ring[i]), lscl->pos_ring_h[i], 0));
    }

    lscl->streams=(hipStream_t *) malloc(sizeof(hipStream_t)*RINGBUF_SIZE);
    for(size_t i=0; i<RINGBUF_SIZE; ++i)
        CHECK(hipStreamCreateWithFlags(lscl->streams+i, 0));

    CHECK(hipDeviceGetAttribute(&lscl->mp_count, hipDeviceAttributeMultiprocessorCount, 0));

    pthread_create(&lscl->getrest, NULL, get_results, (void *) lscl);

    free(buffer);

    return true;
}

void ls_cl_get(ls_cl_t *lscl, const header_t *header) {
    static uint32_t i=0;
#define H(X) lscl->header_ring_h[i][X-1]=header->h ## X
    H(1);
    H(2);
    H(3);
    H(4);
    H(5);
#undef H

    ls<<<1,64,0,lscl->streams[i]>>>(lscl->lower, lscl->upper, (uint64_t) lscl->ruleset->num_rules,
            lscl->header_ring[i], lscl->pos_ring[i]);

    uint8_t stream_running;
    do {
        pthread_mutex_lock(&lscl->running_mtxs[i]);
        stream_running=lscl->streams_running[i];
        pthread_mutex_unlock(&lscl->running_mtxs[i]);
    } while(stream_running);


    pthread_mutex_lock(&lscl->running_mtxs[i]);
    lscl->streams_running[i]=1;
    pthread_mutex_unlock(&lscl->running_mtxs[i]);

    i=(i+1)&RINGBUF_MASK;
}

void ls_cl_free(ls_cl_t *lscl) {
    //lscl->running=0;
    //pthread_join(lscl->getrest, NULL);
    hipFree(lscl->lower);
    hipFree(lscl->upper);
    for(size_t i=0; i<RINGBUF_SIZE; ++i) {
        hipHostFree(lscl->pos_ring_h[i]);
        hipHostFree(lscl->header_ring_h[i]);
    }
    free(lscl->pos_ring);
    free(lscl->header_ring);
    free(lscl->pos_ring_h);
    free(lscl->header_ring_h);
    free(lscl->streams_running);
    free(lscl->streams);
    free(lscl->running_mtxs);
}

int main(int ac, char *as[]) {
    if(ac<3) {
        fprintf(stderr, "Usage: %s [ruleset] [headers] [?result file]\n", as[0]);
        return EXIT_FAILURE;
    }

    FILE *res_file=stdout;
    if(ac>3) {
        if((res_file=fopen(as[3], "w"))==NULL) {
            fprintf(stderr, "could not open \"%s\" for writing!\n", as[3]);
            return EXIT_FAILURE;
        }
    }

    ruleset_t rules= {.num_rules=0, .rules_size=0, .rules=NULL};
    headers_t headers= {.num_headers=0, .headers_size=0, .headers=NULL};
    if(!parse_ruleset(&rules, as[1]) || !parse_headers(&headers, as[2]))
        goto fail;

    struct timeval tv1, tv2;
    ls_cl_t lscl;

    gettimeofday(&tv1, NULL);
    if(!ls_cl_new(&lscl, &rules, res_file)) {
        fputs("could not initiate ls_cl!\n", stderr);
        goto fail;
    }
    gettimeofday(&tv2, NULL);
    printf("PREPROCESSING  took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    gettimeofday(&tv1, NULL);
    for(size_t i=0; i<headers.num_headers; ++i)
        ls_cl_get(&lscl, headers.headers+i);
    lscl.running=0;
    pthread_join(lscl.getrest, NULL);
    gettimeofday(&tv2, NULL);
    printf("CLASSIFICATION took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    ls_cl_free(&lscl);

    return EXIT_SUCCESS;
fail:
    free(rules.rules);
    free(headers.headers);

    return EXIT_FAILURE;
}
