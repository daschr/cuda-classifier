#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <string.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#include "ls_cl.h"
#include "parser.h"
}


static inline void check_error(hipError_t e, const char *file, int line) {
    if(e != hipSuccess) {
        fprintf(stderr, "[ERROR] %s in %s (line %d)\n", hipGetErrorString(e), file, line);
        exit(EXIT_FAILURE);
    }
}
#define CHECK(X) (check_error(X, __FILE__, __LINE__))

static inline void cpy_rules(const ruleset_t *rules, uint32_t *buffer, uint8_t upper) {
    size_t bp;
    for(size_t i=0; i<rules->num_rules; ++i) {
        bp=i<<2;
        buffer[bp++]=rules->rules[i].c1[upper];
        buffer[bp++]=rules->rules[i].c2[upper];
        buffer[bp++]=(uint32_t) (rules->rules[i].c3[upper]<<16) | (uint32_t) rules->rules[i].c4[upper];
        buffer[bp]=rules->rules[i].c5[upper];
    }
}

__global__ void ls(const __restrict__ uint *lower, const __restrict__ uint *upper, const ulong rules_size,
                   const __restrict__ uint *header, uint *pos) {
    ulong i=((blockDim.x*blockIdx.x+threadIdx.x)>>2)<<2, step=(gridDim.x*blockDim.x);
    __shared__ uint8_t found;
    uint8_t r, t=threadIdx.x&3;
    const uint mask=0xf<<((threadIdx.x&31)-t);

    if(!threadIdx.x)
        found=0;

    __syncthreads();
    while(!found) {
        r=t==2?(__vcmpleu2(lower[i+t], header[2]) & __vcmpgeu2(upper[2], header[i+t]))==0xffffffff:
          lower[i+t]<=header[t] & header[t]<=upper[i+t];
        __syncwarp();
        if(__all_sync(mask, r)&(!t)) {
            atomicMin((uint *) pos, i>>2);
            found=1;
        }

        i+=step;

        if(!threadIdx.x)
            if(i>=rules_size)
                found=1;

        __syncthreads();
    }
}

bool ls_cl_new(ls_cl_t *lscl, const ruleset_t *rules) {
    size_t bufsize=(sizeof(uint32_t)<<2)*rules->num_rules;
    uint32_t *buffer=(uint32_t *) malloc(bufsize);
    memset(buffer, 0, bufsize);
    CHECK(hipMalloc((void **) &lscl->lower, bufsize));
    CHECK(hipMalloc((void **) &lscl->upper, bufsize));
    CHECK(hipMalloc((void **) &lscl->header, sizeof(uint32_t)<<2));
    CHECK(hipMalloc((void **) &lscl->pos, sizeof(uint64_t)));

    cpy_rules(rules, buffer, 0);
    CHECK(hipMemcpy(lscl->lower, buffer, bufsize, hipMemcpyHostToDevice));

    cpy_rules(rules, buffer, 1);
    CHECK(hipMemcpy(lscl->upper, buffer, bufsize, hipMemcpyHostToDevice));

    free(buffer);

    CHECK(hipDeviceGetAttribute(&lscl->mp_count, hipDeviceAttributeMultiprocessorCount, 0));

    return true;
}

uint8_t ls_cl_get(ls_cl_t *lscl, const ruleset_t *rules, const header_t *header) {
#define H(X) header->h ## X
    uint32_t h[4]= { H(1), H(2), (uint32_t) (H(3)<<16) | (uint32_t) H(4), H(5) };
#undef H

    CHECK(hipMemcpy(lscl->header, h, sizeof(uint32_t)<<2, hipMemcpyHostToDevice));
    uint64_t p=UINT_MAX;
    CHECK(hipMemcpy(lscl->pos, &p, sizeof(uint32_t), hipMemcpyHostToDevice));

    ls<<<lscl->mp_count,256>>>(lscl->lower, lscl->upper, (uint64_t) rules->num_rules<<2, lscl->header, lscl->pos);

    CHECK(hipMemcpy(&p, lscl->pos, sizeof(uint32_t), hipMemcpyDeviceToHost));
    return p==UINT_MAX?0xff:rules->rules[p].val;
}

void ls_cl_free(ls_cl_t *lscl) {
    hipFree(lscl->lower);
    hipFree(lscl->upper);
    hipFree(lscl->pos);
    hipFree(lscl->header);
}

int main(int ac, char *as[]) {
    if(ac<3) {
        fprintf(stderr, "Usage: %s [ruleset] [headers] [?result file]\n", as[0]);
        return EXIT_FAILURE;
    }

    FILE *res_file=stdout;
    if(ac>3) {
        if((res_file=fopen(as[3], "w"))==NULL) {
            fprintf(stderr, "could not open \"%s\" for writing!\n", as[3]);
            return EXIT_FAILURE;
        }
    }

    ruleset_t rules= {.num_rules=0, .rules_size=0, .rules=NULL};
    headers_t headers= {.num_headers=0, .headers_size=0, .headers=NULL};
    if(!parse_ruleset(&rules, as[1]) || !parse_headers(&headers, as[2]))
        goto fail;

    struct timeval tv1, tv2;
    ls_cl_t lscl;

    gettimeofday(&tv1, NULL);
    if(!ls_cl_new(&lscl, &rules)) {
        fputs("could not initiate ls_cl!\n", stderr);
        goto fail;
    }
    gettimeofday(&tv2, NULL);
    printf("PREPROCESSING  took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    gettimeofday(&tv1, NULL);
    for(size_t i=0; i<headers.num_headers; ++i)
        fprintf(res_file, "%02X\n", ls_cl_get(&lscl, &rules, headers.headers+i));
    gettimeofday(&tv2, NULL);
    printf("CLASSIFICATION took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    ls_cl_free(&lscl);

    return EXIT_SUCCESS;
fail:
    free(rules.rules);
    free(headers.headers);

    return EXIT_FAILURE;
}
