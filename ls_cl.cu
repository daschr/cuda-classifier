#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <string.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#include "ls_cl.h"
#include "parser.h"
}


static inline void check_error(hipError_t e, const char *file, int line) {
    if(e != hipSuccess) {
        fprintf(stderr, "[ERROR] %s in %s (line %d)\n", hipGetErrorString(e), file, line);
        exit(EXIT_FAILURE);
    }
}
#define CHECK(X) (check_error(X, __FILE__, __LINE__))

static inline void cpy_rules(const ruleset_t *rules, uint32_t *buffer, uint8_t upper) {
    size_t bp;
    for(size_t i=0; i<rules->num_rules; ++i) {
        bp=i<<2;
        buffer[bp++]=rules->rules[i].c1[upper];
        buffer[bp++]=rules->rules[i].c2[upper];
        buffer[bp++]=(uint32_t) (rules->rules[i].c3[upper]<<16) | (uint32_t) rules->rules[i].c4[upper];
        buffer[bp]=rules->rules[i].c5[upper];
    }
}

__global__ void ls(const __restrict__ uint *lower, const __restrict__ uint *upper, const ulong rules_size,
                   const __restrict__ uint *header, uint *pos) {
    uint start=(uint) blockDim.x*blockIdx.x+threadIdx.x, step=(uint) (gridDim.x*blockDim.x)<<2;
    __shared__ uint8_t found;
    ulong i=start<<2;
    uint8_t r;

    if(!threadIdx.x)
        found=0;

    __syncthreads();
    while(!found) {
        r=i<rules_size?lower[i]<=header[0] & header[0]<=upper[i]
          & lower[i+1]<=header[1] & header[1]<=upper[i+1]
          & (__vcmpleu2(lower[i+2], header[2]) & __vcmpgeu2(upper[i+2], header[2]))==0xffffffff
          & lower[i+3]<=header[3] & header[3]<=upper[i+3]:0;

        if(r) {
            atomicMin((uint *) pos, i>>2);
            found=1;
        }

        if((!threadIdx.x) & (i>=rules_size))
            found=1;

        i+=step;
        __syncthreads();
    }
}

bool ls_cl_new(ls_cl_t *lscl, const ruleset_t *rules) {
    size_t bufsize=(sizeof(uint32_t)<<2)*rules->num_rules;
    uint32_t *buffer=(uint32_t *) malloc(bufsize);
    memset(buffer, 0, bufsize);
    CHECK(hipMalloc((void **) &lscl->lower, bufsize));
    CHECK(hipMalloc((void **) &lscl->upper, bufsize));
    CHECK(hipMalloc((void **) &lscl->header, sizeof(uint32_t)<<2));
    CHECK(hipMalloc((void **) &lscl->pos, sizeof(uint64_t)));

    cpy_rules(rules, buffer, 0);
    CHECK(hipMemcpy(lscl->lower, buffer, bufsize, hipMemcpyHostToDevice));

    cpy_rules(rules, buffer, 1);
    CHECK(hipMemcpy(lscl->upper, buffer, bufsize, hipMemcpyHostToDevice));

    free(buffer);

    CHECK(hipDeviceGetAttribute(&lscl->mp_count, hipDeviceAttributeMultiprocessorCount, 0));

    return true;
}

uint8_t ls_cl_get(ls_cl_t *lscl, const ruleset_t *rules, const header_t *header) {
#define H(X) header->h ## X
    uint32_t h[4]= { H(1), H(2), (uint32_t) (H(3)<<16) | (uint32_t) H(4), H(5) };
#undef H

    CHECK(hipMemcpy(lscl->header, h, sizeof(uint32_t)<<2, hipMemcpyHostToDevice));
    uint64_t p=UINT_MAX;
    CHECK(hipMemcpy(lscl->pos, &p, sizeof(uint32_t), hipMemcpyHostToDevice));

    ls<<<1,128>>>(lscl->lower, lscl->upper, (uint64_t) rules->num_rules<<2, lscl->header, lscl->pos);

    CHECK(hipMemcpy(&p, lscl->pos, sizeof(uint32_t), hipMemcpyDeviceToHost));
    return p==UINT_MAX?0xff:rules->rules[p].val;
}

void ls_cl_free(ls_cl_t *lscl) {
    hipFree(lscl->lower);
    hipFree(lscl->upper);
    hipFree(lscl->pos);
    hipFree(lscl->header);
}

int main(int ac, char *as[]) {
    if(ac<3) {
        fprintf(stderr, "Usage: %s [ruleset] [headers] [?result file]\n", as[0]);
        return EXIT_FAILURE;
    }

    FILE *res_file=stdout;
    if(ac>3) {
        if((res_file=fopen(as[3], "w"))==NULL) {
            fprintf(stderr, "could not open \"%s\" for writing!\n", as[3]);
            return EXIT_FAILURE;
        }
    }

    ruleset_t rules= {.num_rules=0, .rules_size=0, .rules=NULL};
    headers_t headers= {.num_headers=0, .headers_size=0, .headers=NULL};
    if(!parse_ruleset(&rules, as[1]) || !parse_headers(&headers, as[2]))
        goto fail;

    struct timeval tv1, tv2;
    ls_cl_t lscl;

    gettimeofday(&tv1, NULL);
    if(!ls_cl_new(&lscl, &rules)) {
        fputs("could not initiate ls_cl!\n", stderr);
        goto fail;
    }
    gettimeofday(&tv2, NULL);
    printf("PREPROCESSING  took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    gettimeofday(&tv1, NULL);
    for(size_t i=0; i<headers.num_headers; ++i)
        fprintf(res_file, "%02X\n", ls_cl_get(&lscl, &rules, headers.headers+i));
    gettimeofday(&tv2, NULL);
    printf("CLASSIFICATION took %12lu us\n", 1000000*(tv2.tv_sec-tv1.tv_sec)+(tv2.tv_usec-tv1.tv_usec));

    ls_cl_free(&lscl);

    return EXIT_SUCCESS;
fail:
    free(rules.rules);
    free(headers.headers);

    return EXIT_FAILURE;
}
